#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include <cusolver_utils.h>

namespace ridge
{
    std::vector<double> solve(std::vector<double> const &rhs_matrix, std::vector<double> const &lhs_vector, int const m, int const n, double const lamda)
    {
        hipsolverHandle_t cusolverH = NULL;
        hipblasHandle_t cublasH = NULL;
        hipStream_t stream{};

        // create lamda diagonal matrix
        std::vector<double> rhs_t_rhs_prod(n * n, 0);

        // create data for solution
        std::vector<double> x(n, 0);

        // device/gpu memory pointers
        double *d_rhs_matrix = nullptr;
        double *d_rhs_t_rhs_prod = nullptr;
        double *d_lamda_diagonal = nullptr;
        double *d_tau = nullptr;
        double *d_lhs_vector = nullptr;
        double *d_rhs_t_lhs_prod = nullptr;
        int *d_info = nullptr;
        double *d_work = nullptr;

        int lwork_geqrf = 0;
        int lwork_ormqr = 0;
        int lwork = 0;
        int info = 0;

        double const one = 1;
        int const nrhs = 1;
        // create handlers
        CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
        CUBLAS_CHECK(hipblasCreate(&cublasH));

        CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
        CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));
        CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

        // gpu memory allocation and copying of data
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_rhs_matrix), sizeof(double) * rhs_matrix.size()));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_rhs_t_rhs_prod), sizeof(double) * n * n));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_tau), sizeof(double) * n));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_lhs_vector), sizeof(double) * lhs_vector.size()));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_rhs_t_lhs_prod), sizeof(double) * n));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

        CUDA_CHECK(hipMemcpyAsync(d_rhs_matrix, rhs_matrix.data(), sizeof(double) * rhs_matrix.size(), hipMemcpyHostToDevice, stream));
        CUDA_CHECK(hipMemcpyAsync(d_lhs_vector, lhs_vector.data(), sizeof(double) * lhs_vector.size(), hipMemcpyHostToDevice, stream));

        // compute intermediate computations before qr algorithm
        // variables needed for multiplications
        hipblasOperation_t transpose = HIPBLAS_OP_T;
        hipblasOperation_t no_transpose = HIPBLAS_OP_N;
        double const alpha = 1.0;
        double const beta = 0.0;
        // compute rhs_matrix^T * rhs_matrix + d_lamda_diagonal = d_lamda_diagonal
        int const lda = n;
        int const ldb = n;
        int const ldc = n;
        CUBLAS_CHECK(hipblasDgemm(cublasH, transpose, no_transpose, m, n, n, &alpha, d_rhs_matrix, lda, d_rhs_matrix, ldb, &beta, d_rhs_t_rhs_prod, ldc));

        // to add lamda_diagonal, copy the data into cpu memory and add lamda on diagonal and then copy it back into gpu memory
        CUDA_CHECK(hipMemcpyAsync(rhs_t_rhs_prod.data(), d_rhs_t_rhs_prod, sizeof(double) * rhs_t_rhs_prod.size(), hipMemcpyDeviceToHost, stream));
        for (int i = 0; i < n; i++)
        {
            rhs_t_rhs_prod[i * (n + 1)] += lamda;
        }
        CUDA_CHECK(hipMemcpyAsync(d_rhs_t_rhs_prod, rhs_t_rhs_prod.data(), sizeof(double) * rhs_t_rhs_prod.size(), hipMemcpyHostToDevice, stream));

        // compute rhs_matrix^T * lhs_vector = d_rhs_t_lhs_prod
        int const incx = 1;
        int const incy = 1;
        CUBLAS_CHECK(hipblasDgemv(cublasH, transpose, m, n, &alpha, d_rhs_matrix, lda, d_lhs_vector, incx, &beta, d_rhs_t_lhs_prod, incy));

        // free unused memory
        CUDA_CHECK(hipFree(d_rhs_matrix));
        CUDA_CHECK(hipFree(d_lhs_vector));

        // query working space of geqrf and ormqr
        CUSOLVER_CHECK(hipsolverDnDgeqrf_bufferSize(cusolverH, m, m, d_A, lda, &lwork_geqrf));

        CUSOLVER_CHECK(hipsolverDnDormqr_bufferSize(cusolverH, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, m, nrhs, m,
                                                   d_A, lda, d_tau, d_B, ldb, &lwork_ormqr));

        lwork = std::max(lwork_geqrf, lwork_ormqr);

        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(double) * lwork));

        // compute qr of d_lamda_diagonal and check if succeded
        CUSOLVER_CHECK(hipsolverDnDgeqrf(cusolverH, n, n, d_rhs_t_rhs_prod, lda, d_tau, d_work, lwork, d_info));

        CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

        CUDA_CHECK(hipStreamSynchronize(stream));

        std::printf("after geqrf: info = %d\n", info);
        if (0 > info)
        {
            std::printf("%d-th parameter is wrong \n", -info);
            exit(1);
        }

        // compute Q^T * d_rhs_t_lhs_prod
        CUSOLVER_CHECK(hipsolverDnDormqr(cusolverH, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, n, nrhs, n, d_rhs_t_rhs_prod, lda,
                                        d_tau, d_rhs_t_lhs_prod, ldb, d_work, lwork, d_info));

        /* check if QR is good or not */
        CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

        CUDA_CHECK(hipStreamSynchronize(stream));

        std::printf("after ormqr: info = %d\n", info);
        if (0 > info)
        {
            std::printf("%d-th parameter is wrong \n", -info);
            exit(1);
        }

        // solve QRx = d_rhs_t_lhs_prod
        CUBLAS_CHECK(hipblasDtrsm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N,
                                 HIPBLAS_DIAG_NON_UNIT, n, nrhs, &one, d_rhs_t_rhs_prod, lda, d_rhs_t_lhs_prod, ldb));

        CUDA_CHECK(hipMemcpyAsync(x.data(), d_rhs_t_lhs_prod, sizeof(double) * x.size(), hipMemcpyDeviceToHost,
                                   stream));

        CUDA_CHECK(hipStreamSynchronize(stream));

        // free resources
        CUDA_CHECK(hipFree(d_rhs_t_rhs_prod));
        CUDA_CHECK(hipFree(d_tau));
        CUDA_CHECK(hipFree(d_rhs_t_lhs_prod));
        CUDA_CHECK(hipFree(d_info));
        CUDA_CHECK(hipFree(d_work));

        CUBLAS_CHECK(hipblasDestroy(cublasH));
        CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

        CUDA_CHECK(hipStreamDestroy(stream));

        CUDA_CHECK(hipDeviceReset());
    }
}